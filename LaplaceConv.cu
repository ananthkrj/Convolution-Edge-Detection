#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

/**
 * Define compile time constants, will need for shared memory,
 * halo regions, and shared memory tile size
*/
#define TILE_SIZE 16
#define HALO_SIZE 1
#define SHARED_SIZE (TILE_SIZE + 2 * HALO_SIZE)

// Macro for checking CUDA runtime calls
// Macro needs do while loop because this makes sure contents inside
// do while loop always behave the same regardless of how
//#define CUDA_CHECK(call) \
   // do { \
       // hipError_t err = call; \
       // if (err != hipSuccess) { \
            //fprintf(stderr, "Cuda error present at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
           // exit(EXIT_FAILURE); \
       // } \
   // } while(0)

// Macro For checking kernel launches
//#define CUDA_KERNEL_LAUNCH() \
   // do { \
        hipError_t err = hipGetLastError(); \
        if (err != hipSuccess) { \
            fprintf(stderr, "Cuda error present at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
        err = hipDeviceSynchronize(); \
        if (err != hipSuccess) { \
            fprintf(stderr, "Cuda error present at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)



// Laplacian Kernel
// width and height are the dimensons of the image
__global__ void LaplacianKernel(float* input, float* output, int width, int height) {
    // Load constants and Halo info into shared memory
    __shared__ float shared_data[SHARED_SIZE][SHARED_SIZE];

    // initialize row and col using thread and block
    // (Reminder it is 2 dimensions)
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute shared memory coordinates (where to store the shared memory)
    int ty = threadIdx.y + HALO_SIZE;
    int tx = threadIdx.x + HALO_SIZE;

    // Apply the 3x3 laplacian kernel, (consists of calculating halo regions)
    // Implement like: left, right, top, bottom
    
    // load main tile into shared memory
    // double check if it is actuall row < height && col < width later
    if (row < height && col < width) {
        shared_data[ty][tx] = input[row * width + col];
    } else {
        // populate with 0's if out of bounds
        shared_data[ty][tx] = 0.0f;
    }
    
    // calculate for left first
    // leftmost is column index
    if (threadIdx.x == 0) {
        int Halo_Col = col - HALO_SIZE;
        // set bounds for calculating left
        if (Halo_Col >= 0 && row < height) {
            shared_data[ty][tx - HALO_SIZE] = input[row * width + Halo_Col];
        } else {
            shared_data[ty][tx - HALO_SIZE] = 0.0f;
        }
    }

    // calculate for right
    // rightmost is column index to right of main tile
    if (threadIdx.x == blockDim.x - 1) {
        // new halo col calculation
        int Halo_Col = col + HALO_SIZE;
        // bounds check for right
        if (Halo_Col < width && row < height) {
            // load right tile in shared memory
            shared_data[ty][tx + HALO_SIZE] = input[row * width + Halo_Col];
        } else {
            shared_data[ty][tx + HALO_SIZE] = 0.0f;
        }
    }

    // calculate for top and bottom now

    // check if top exists
    if (threadIdx.y == 0) {
        int Halo_Row = row - HALO_SIZE;
        // in bounds
        if (Halo_Row >= 0 && col < width) {
            // load into shared memoy
            shared_data[ty - HALO_SIZE][tx] = input[Halo_Row * width + col];
        } else {
            shared_data[ty - HALO_SIZE][tx] = 0.0f;
        }
    }

    // Calculate for bottom now
    if (threadIdx.y == blockDim.y - 1) {
        int Halo_Row = row + HALO_SIZE;
        // bounds check to populate
        if (Halo_Row < height && col < width) {
            shared_data[ty + HALO_SIZE][tx] = input[Halo_Row * width + col];
        } else {
            shared_data[ty + HALO_SIZE][tx] = 0.0f;
        }
    }

    // sync all threads
    __syncthreads();

    // then compute laplacian convolution using HALO's
    // check if we are in range first
    // check if this is the correct calculation
    if (row < height && col < width) {
        // store computation in result variable
        // add all shared memory points, main tile + top + bottom + left + right
        // error1: fix calc
        float result = 4.0f * shared_data[ty][tx] - shared_data[ty - 1][tx] - shared_data[ty + 1][tx] -
                       shared_data[ty][tx - 1] - shared_data[ty][tx + 1];
        

        // store result in output array
        output[row * width + col] = result;
    }

    // synch all shared memory (final)?
    // Not sure if i need this, probably not as syncthreads
    // likely only needed after halo's loaded into shared mem
    __syncthreads();
}

/**
 * Cuda kernel launch helper
 * represent input and output messages as mat arrays. Pass them by address
 * Will mainly contain device memory implementation
 * inputImg and outputImg are the host memory
*/
void runLaplacian(cv::Mat& inputImg, cv::Mat& outputImg) {
    // Calculate Memory Size
    size_t size = inputImg.rows * inputImg.cols * sizeof(float);

    // Allocate Device memory using hipMalloc
    float *d_input;
    hipError_t err = hipMalloc(&d_input, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc d_input failed: %s\n", hipGetErrorString(err));
        return;
    }
    
    float *d_output;
    err = hipMalloc(&d_output, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc d_output failed: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        return;
    }

    // Copy from host to device
    err = hipMemcpy(d_input, inputImg.ptr<float>(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy H2D failed: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // allocate block and grid dimensions
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((inputImg.cols + TILE_SIZE - 1) / TILE_SIZE, 
                (inputImg.rows + TILE_SIZE - 1) / TILE_SIZE);

    printf("Image size: %dx%d\n", inputImg.cols, inputImg.rows);
    printf("Grid size: %dx%d, Block size: %dx%d\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

    // Launch kernel
    LaplacianKernel<<<gridDim, blockDim>>>(d_input, d_output, inputImg.cols, inputImg.rows);
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return;
    }
    
    // Wait for kernel to finish and check for execution errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    printf("Kernel executed successfully!\n");

    // Copy from device to host
    err = hipMemcpy(outputImg.ptr<float>(), d_output, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy D2H failed: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    
    printf("Memory operations completed successfully!\n");
}

/**
 * Work on main before the helper function to better understand how helper works
 * Main function:
 * Will be used for cv image processing and cv intertwined host memory 
 * Will utilize command line arguments
*/
int main(int argc, char** argv) {
    std::string imagePath;
    
    // Use bundled lena.jpg if no argument provided
    if (argc == 2) {
        imagePath = argv[1];
    } else {
        imagePath = "Lena.jpg";
        std::cout << "Using bundled test image: " << imagePath << std::endl;
    }

    // Read and open image as grayscale
    cv::Mat img = cv::imread(imagePath, cv::IMREAD_GRAYSCALE);

    // Check if image loaded successfully
    if (img.empty()) {
        std::cerr << "Error: Could not load image: " << imagePath << std::endl;
        return -1;
    }

    // Convert to float32 and normalize to [0, 1]
    cv::Mat imgFloat;
    img.convertTo(imgFloat, CV_32F, 1.0/255.0);

    // Create output matrix
    cv::Mat output(img.rows, img.cols, CV_32F);

    // Run CUDA Laplacian kernel
    runLaplacian(imgFloat, output);

    // Convert output back to 8-bit for saving
    cv::Mat outputUint8;
    cv::Mat absOutput;
    cv::absdiff(output, cv::Scalar::all(0), absOutput);
    
    double minVal, maxVal;
    cv::minMaxLoc(absOutput, &minVal, &maxVal);
    
    if (maxVal > 0) {
        absOutput.convertTo(outputUint8, CV_8U, 255.0/maxVal);
    } else {
        outputUint8 = cv::Mat::zeros(img.rows, img.cols, CV_8U);
    }

    // Save the result image
    if (!cv::imwrite("laplacian_result.jpg", outputUint8)) {
        std::cerr << "Error saving image to file" << std::endl;
        return -1;
    }

    std::cout << "Laplacian edge detection completed! Output saved as laplacian_result.jpg" << std::endl;
    return 0;
}

